#include "CUDAComputation.h"
#include "CUDAkernel.cuh"
#include <iostream>
#include <chrono>

template <typename T>
CUDAComputation<T>::CUDAComputation(const Matrix<T> &a, const Matrix<T> &b, const Matrix<T> &c, const Matrix<T> &d, const Matrix<T> &e, Matrix<T> &result)
    : Computation<T>(a, b, c, d, e, result)
{
    bufferSize = this->width * this->height * sizeof(T);
}

template <typename T>
void CUDAComputation<T>::init(int device_ind)
{
    selectDevice(device_ind);
}

template <typename T>
std::vector<hipDeviceProp_t> CUDAComputation<T>::getAvailableDevices()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    std::vector<hipDeviceProp_t> devices(deviceCount);
    for (int i = 0; i < deviceCount; ++i)
    {
        hipGetDeviceProperties(&devices[i], i);
    }
    return devices;
}

template <typename T>
void CUDAComputation<T>::printDevices(const std::vector<hipDeviceProp_t> &devices)
{
    std::cout << "Available CUDA devices:" << std::endl;
    for (size_t i = 0; i < devices.size(); ++i)
    {
        std::cout << "Device " << i << ": " << devices[i].name << std::endl;
    }
}

template <typename T>
void CUDAComputation<T>::selectDevice(int index)
{
    std::vector<hipDeviceProp_t> devices = getAvailableDevices();
    if (index < 0 || index >= devices.size())
    {
        throw std::runtime_error("Invalid device index.");
    }
    deviceIndex = index;
    hipSetDevice(deviceIndex);
    std::cout << "Using CUDA device: " << devices[deviceIndex].name << std::endl;
}

template <typename T>
void CUDAComputation<T>::initBuffers()
{
    hipMalloc(&d_a, bufferSize);
    hipMalloc(&d_b, bufferSize);
    hipMalloc(&d_c, bufferSize);
    hipMalloc(&d_d, bufferSize);
    hipMalloc(&d_e, bufferSize);
    hipMalloc(&d_result, bufferSize);
}

template <typename T>
void CUDAComputation<T>::freeBuffers()
{
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
    hipFree(d_e);
    hipFree(d_result);
}

template <typename T>
std::chrono::duration<double, std::milli> CUDAComputation<T>::run()
{
    initBuffers();

    hipMemcpy(d_a, this->a.data.data(), bufferSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, this->b.data.data(), bufferSize, hipMemcpyHostToDevice);
    hipMemcpy(d_c, this->c.data.data(), bufferSize, hipMemcpyHostToDevice);
    hipMemcpy(d_d, this->d.data.data(), bufferSize, hipMemcpyHostToDevice);
    hipMemcpy(d_e, this->e.data.data(), bufferSize, hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 gridSize((this->width + blockSize.x - 1) / blockSize.x, (this->height + blockSize.y - 1) / blockSize.y);

    auto computeStart = std::chrono::high_resolution_clock::now();
    matrixCompute<<<gridSize, blockSize>>>(d_a, d_b, d_c, d_d, d_e, d_result, this->width, this->height);
    hipDeviceSynchronize();
    auto computeEnd = std::chrono::high_resolution_clock::now();

    hipMemcpy(this->result.data.data(), d_result, bufferSize, hipMemcpyDeviceToHost);

    freeBuffers();

    return computeEnd - computeStart;
}

template class CUDAComputation<float>;
template class CUDAComputation<double>;
